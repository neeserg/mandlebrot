
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

__global__
void add_with_thread(int n, float *x, float *y)
{
  int index = threadIdx.x;
  int stride = blockDim.x;
  for (int i = index; i < n; i += stride)
      y[i] = x[i] + y[i];
}
int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  gpuErrchk(hipMallocManaged(&x, N*sizeof(float)));
  gpuErrchk(hipMallocManaged(&y, N*sizeof(float)));
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  //add<<<numBlocks, blockSize>>>(N, x, y);
  add_with_thread<<<1,256>>>(N, x, y);
  // Run kernel on 1M elements on the GPU

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
  std::cout << "Block Dimensions: " << numBlocks << std::endl;
    std::cout << "Dimensions: " << N << std::endl;


  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}